#include "hip/hip_runtime.h"
#include "BufferConverter_Stereo32_CUDA.h"

__global__ void toPCMStereo32_kernel(const float* bufferL, const float* bufferR, uint8_t* output, const uint sampleSize){
    const uint maxValue = 0x7FFFFF80;
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < sampleSize){
        int valueL = maxValue * bufferL[i];
        int valueR = maxValue * bufferR[i];
        i *= 8;
        output[i++] = valueL;
        output[i++] = valueL >> 8;
        output[i++] = valueL >> 16;
        output[i++] = valueL >> 24;
        output[i++] = valueR;
        output[i++] = valueR >> 8;
        output[i++] = valueR >> 16;
        output[i] = valueR >> 24;
    }
}

BufferConverter_Stereo32_CUDA::BufferConverter_Stereo32_CUDA(const uint& sampleSize) : sampleSize(sampleSize){
    hipMalloc((void**)(&d_buffer), sampleSize*8);
}

BufferConverter_Stereo32_CUDA::~BufferConverter_Stereo32_CUDA(){
    hipFree(this->d_buffer);
}

void BufferConverter_Stereo32_CUDA::toPCM(pipelineAudioBuffer_CUDA* pipelineBuffer, audioBuffer* pcmBuffer){
    uint blockCount = (sampleSize + CUDA_BUFFER_CONVERTER_BLOCK_SIZE - 1) / CUDA_BUFFER_CONVERTER_BLOCK_SIZE;
    toPCMStereo32_kernel<<<blockCount, CUDA_BUFFER_CONVERTER_BLOCK_SIZE>>>(pipelineBuffer->d_bufferL, pipelineBuffer->d_bufferR, d_buffer, sampleSize);
    hipDeviceSynchronize();
    hipMemcpy(pcmBuffer->buff, d_buffer, sampleSize*8, hipMemcpyDeviceToHost);
}