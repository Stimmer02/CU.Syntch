#include "hip/hip_runtime.h"
#include "BufferConverter_Mono32_CUDA.h"

__global__ void toPCMMono32_kernel(const float* bufferL, const float* bufferR, uint8_t* output, const uint sampleSize){
    const uint maxValue = 0x7FFFFF80;
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < sampleSize){
        int value = maxValue * bufferL[i];
        // int value = maxValue * (bufferL[i] + bufferR[i]) / 2;
        i *= 4;
        output[i++] = value;
        output[i++] = value >> 8;
        output[i++] = value >> 16;
        output[i] = value >> 24;
    }
}

BufferConverter_Mono32_CUDA::BufferConverter_Mono32_CUDA(const uint& sampleSize) : sampleSize(sampleSize){
    hipMalloc((void**)(&d_buffer), sampleSize*4);
}

BufferConverter_Mono32_CUDA::~BufferConverter_Mono32_CUDA(){
    hipFree(this->d_buffer);
}

void BufferConverter_Mono32_CUDA::toPCM(pipelineAudioBuffer_CUDA* pipelineBuffer, audioBuffer* pcmBuffer){
    uint blockCount = (sampleSize + CUDA_BUFFER_CONVERTER_BLOCK_SIZE - 1) / CUDA_BUFFER_CONVERTER_BLOCK_SIZE;
    toPCMMono32_kernel<<<blockCount, CUDA_BUFFER_CONVERTER_BLOCK_SIZE>>>(pipelineBuffer->d_bufferL, pipelineBuffer->d_bufferR, d_buffer, sampleSize);
    hipDeviceSynchronize();
    hipMemcpy(pcmBuffer->buff, d_buffer, sampleSize*4, hipMemcpyDeviceToHost);
}
