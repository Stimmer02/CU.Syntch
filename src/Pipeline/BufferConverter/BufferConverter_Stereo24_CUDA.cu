#include "hip/hip_runtime.h"
#include "BufferConverter_Stereo24_CUDA.h"

__global__ void BufferConverter_Stereo24_CUDA::toPCM_kernel(const float* bufferL, const float* bufferR){
    const uint maxValue = 0x007FFFFF;
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < sampleSize){
        int valueL = maxValue * bufferL[i];
        int valueR = maxValue * bufferR[i];
        i *= 6;
        d_buffer[i++] = valueL;
        d_buffer[i++] = valueL >> 8;
        d_buffer[i++] = valueL >> 16;
        d_buffer[i++] = valueR;
        d_buffer[i++] = valueR >> 8;
        d_buffer[i] = valueR >> 16;
    }
}
