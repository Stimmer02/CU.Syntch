#include "hip/hip_runtime.h"
#include "BufferConverter_Stereo16_CUDA.h"

__global__ void BufferConverter_Stereo16_CUDA::toPCM_kernel(const float* bufferL, const float* bufferR){
    const uint maxValue = 0x00007FFF;
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < sampleSize){
        int valueL = maxValue * bufferL[i];
        int valueR = maxValue * bufferR[i];
        i *= 4;
        d_buffer[i++] = valueL;
        d_buffer[i++] = valueL >> 8;
        d_buffer[i++] = valueR;
        d_buffer[i] = valueR >> 8;
    }
}
